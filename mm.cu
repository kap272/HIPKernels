
#include <hip/hip_runtime.h>
#include<assert.h>
#include<stdio.h>
#ifdef __HIPCC__
#include <hip/amd_detail/amd_hip_fp8.h>
#include <hip/amd_detail/amd_hip_bf16.h>
#include <hip/hip_runtime.h>
#endif



// get C_ij where
// C := A @ B_T
// C_ij = A_row_i * B_T_col_j
//      = A_row_i * B_row_j
// A m by k
// B n by k
// both column major format
float get_cell_value(float* A, float* B, int m, int k, int n) {

} 


float* generate_random_vector(int n) {
    float* vec = (float*) malloc(sizeof(float) * n); 
    for (int i = 0; i < n; i++) {
        vec[i] = rand() % 10;
    }
    return vec;
}

float* allocate_matrix(int rows, int cols) {
    float* vec = (float*) malloc(sizeof(float) * rows * cols); 
    return vec;
}

float* generate_random_matrix(int rows, int cols) {
    float* M = (float*) malloc(sizeof(float) * rows * cols); 
    for (int i = 0; i < rows * cols; i++) {
        M[i] = rand() % 10;
    }
    return M;

}

void print_vector(float* v, int n) {
    for (int i = 0; i < n; i++) {
        printf("%f\n", v[i]);
    }
    printf("\n");
}    

void print_matrix(float* M, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            printf("%f ", M[cols * i + j]);
        }
        printf("\n");
    }
    printf("\n");
}

bool matrices_are_equal(float* A, float *B, int A_rows, int A_cols, int B_rows, int B_cols) {
    if (A_rows != B_rows || A_cols != B_cols) {
        return false;
    }

    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < A_cols; j++) {
            //if (A[i * A_cols + j] != B[i * B_cols + j]) {
            if (A[j * A_cols + i] != B[j * B_cols + i]) {
                return false;
            }
        }
    }

    return true;

}

void mat_mul(float* A, float* B, float* C, int A_rows, int A_cols, int B_cols) {
    for (int i = 0; i < A_rows; i++) {
        for (int j = 0; j < B_cols; j++) {
            float C_val = 0.0;
            for (int l = 0; l < A_cols; l++) {
                C_val += A[i * A_cols + l] * B[l * B_cols + j];
            }
            C[i * B_cols + j] = C_val;
        }
    }
}


typedef void (*mat_mul_func)(const float* a, const float* b, const float* as, const float* bs, float* c, int m, int n, int k); 

void cudaMalloc_wrapped(float* addr, int size) {

}

void cudaMemcpy_wrapped(float* addr, int size) {

}

void cudaDeviceSynchronize_wrapped() {
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel execution failed: %s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

float compare_mat_mul_kernels(mat_mul_func f, mat_mul_func g, int m, int k, int n, bool print_results, dim3 f_grid_dims, dim3 f_block_dims, dim3 g_grid_dims, dim3 g_block_dims) {
    // A m-by-k matrix in column major format
    // B k-by-n matrix in column major format (repressented as a n-by-k)
    // C m-by-n matrix in row major format
    int A_size = sizeof(float) * m * k;
    int B_size = sizeof(float) * n * k;
    int C_size = sizeof(float) * m * n;
    int alpha_size = sizeof(float) * m * k/128;
    int beta_size = sizeof(float) * n * k/128;

    float* A = generate_random_matrix(m, k); 
    float* B = generate_random_matrix(n, k); 


    float* alpha = generate_random_matrix(m, k/128);
    float* beta = generate_random_matrix(n, k/128);


    // establish ground truth
    // print_matrix(A, A_rows, A_cols);
    // print_matrix(B, A_cols, B_cols);

    // run kernel f
    float* C_h_f = allocate_matrix(m, n);

    float* A_d;
    float* B_d;
    float* alpha_d;
    float* beta_d;
    float* C_d_f = allocate_matrix(m, n);

    hipMalloc(&A_d, A_size);
    hipMalloc(&B_d, B_size); 
    hipMalloc(&C_d_f, C_size); 
    hipMalloc(&alpha_d, alpha_size); 
    hipMalloc(&beta_d, beta_size); 

    hipMemcpy(A_d, A, A_size, hipMemcpyHostToDevice); 
    hipMemcpy(B_d, B, B_size, hipMemcpyHostToDevice); 
    hipMemcpy(alpha_d, alpha, alpha_size, hipMemcpyHostToDevice); 
    hipMemcpy(beta_d, beta, beta_size, hipMemcpyHostToDevice); 

    float f_time;
    hipEvent_t f_start, f_stop;    
    hipEventCreate(&f_start);
    hipEventCreate(&f_stop);
    hipEventRecord(f_start, 0);

    // TODO: a better way to determine kernel launch dimensions
    f<<<f_grid_dims, f_block_dims>>>(A_d, B_d, alpha_d, beta_d, C_d_f, m, n, k);
    
    hipEventRecord(f_stop, 0);
    cudaDeviceSynchronize_wrapped();
    hipEventElapsedTime(&f_time, f_start, f_stop);

    hipMemcpy(C_h_f, C_d_f, C_size, hipMemcpyDeviceToHost); 
    hipFree(C_d_f);

    // then knl g
    float* C_h_g = (float*) malloc(sizeof(float) * m * n); 
    float* C_d_g = (float*) malloc(sizeof(float) * m  * n); 
    printf("%p %p \n", &C_h_g, &C_h_f); //0x7ffeee88cf78 0x7ffeee88cf38
    hipMalloc(&C_d_g, C_size);

    // TODO fix timing
    float g_time;
    hipEvent_t g_start, g_stop;    
    hipEventCreate(&g_start);
    hipEventCreate(&g_stop);
    hipEventRecord(g_start, 0);

    // TODO: a better way to determine kernel launch dimensions
    g<<<g_grid_dims, g_block_dims>>>(A_d, B_d, alpha_d, beta_d, C_d_g, m, n, k);

    hipEventRecord(g_stop, 0);
    cudaDeviceSynchronize_wrapped();
    hipEventElapsedTime(&g_time, g_start, g_stop);

    hipMemcpy(C_h_g, C_d_g, C_size, hipMemcpyDeviceToHost); 

    if (matrices_are_equal(C_h_f, C_h_g, m, n, m, n)) {
        printf("results match \n");
    } else {
        printf("results dont match \n");
    }
    

    if (!matrices_are_equal(C_h_f, C_h_g, m, n, m, n)) {
        printf("kernel g is wrong!\n");
    }
    hipFree(C_d_g);
    if (print_results) {
        int R = 0;
        printf("g result: \n");

        for (int row = 0; row < R; row++) {
            for (int col = 0; col < R; col++) {
                printf("%f ", C_h_g[row * n + col]);
            }
            printf("\n");
        }

        printf("f result: \n");

        for (int row = 0; row < R; row++) {
            for (int col = 0; col < R; col++) {
                printf("%f ", C_h_f[row * n + col]);
            }
            printf("\n");
        }


        int miss = 0;
        int total = 0;
        int zero = 0;
        for (int row = 0; row < m; row++) {
            for (int col = 0; col < n; col++) {
                if (C_h_f[row * n + col] != C_h_g[row * n + col]) {
                   miss += 1; 
                   if (C_h_g[row * n + col] == 0.0) {
                       zero += 1;
                   }
                }
                total += 1;
            }
        }
        printf("miss: %d zero: %d total: %d \n", miss, zero, total);
        printf("miss rate : %f \n", miss/(float)total);
        printf("zero rate : %f \n ", zero/(float)total);
    }

    printf("kernel f run time: %f\n", f_time);
    printf("kernel g run time: %f\n", g_time);
    hipFree(A_d);
    hipFree(B_d);
    free(A);
    free(B);

    return ((g_time - f_time)/f_time);
}
constexpr const int BLOCK = 128;
constexpr const int TILE_WIDTH = 16;
constexpr const int TILES_PER_BLOCK = BLOCK/TILE_WIDTH;

// A, B, C, as, bs, C, m, n, k
__global__ void mat_mul_ref(const float* a, const float* b, const float* as, const float* bs, float* c, int m, int n, int k) {
                   
    // Your implementation here
    int cx = threadIdx.x + blockDim.x * blockIdx.x;
    int cy = threadIdx.y + blockDim.y * blockIdx.y;
    if(cx >= m || cy >= n) return;
    
    int sn = (n + BLOCK - 1) / BLOCK;
    
    float result = 0;
    // split loop into an outer loop over different blocks, and an inner loop within one block.
    // we can assume k % BLOCK == 0.
    for(int i = 0; i < k; i += BLOCK) {
        // block results accumulates the inner product across a single block.
        // within each block, scales are constant, so we can lift the scaling 
        // outside of the inner loop.
        float block_result = 0;
        for(int ii = 0; ii < BLOCK; ++ii) {
            // load input matrix elements and convert to float for computations
            float av = (float)a[cx + (i + ii) * m];
            float bv = (float)b[cy + (i + ii) * n];
            block_result += av * bv; 
        }
        
        // before we can go to the next block, scale the result of the current block
        // and accumulate to final result
        // note the different indexing into as and bs
        result += block_result ;//* as[cx + i/BLOCK * m] * bs[cy/BLOCK + i/BLOCK * sn];
    }
    
    // finally, write the result as bf16
    c[cx * n + cy] = result; 
}

__global__ void custom_kernel_bak(const float* a, const float* b, const float* as, const float* bs, float* c, int m, int n, int k) {
    // Your implementation here
    int row = threadIdx.x + blockDim.x * blockIdx.x;
    int col = threadIdx.y + blockDim.y * blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    if(row >= m || col >= n) {
        return;
    };
    
    int sn = (n + BLOCK - 1) / BLOCK;
    
    __shared__ float A_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_tile[TILE_WIDTH][TILE_WIDTH];

    float result = 0.0;

    // go across the inner dimension (horizonatlly across A and horizontally across B in increments of BLOCK

    for (int inner_dim_idx = 0; inner_dim_idx < k; inner_dim_idx += BLOCK) {
        float block_result = 0.0;
        for (int tile_num = 0; tile_num < TILES_PER_BLOCK; tile_num++) {
            A_tile[tx][ty] = a[row + m * (inner_dim_idx  + (tile_num * TILE_WIDTH) + ty)];
            B_tile[ty][tx] = b[col + n * (inner_dim_idx  + (tile_num * TILE_WIDTH )+ tx)];
            __syncthreads();
            for (int k = 0; k < TILE_WIDTH; k++) {
                block_result += A_tile[tx][k] * B_tile[ty][k];
            }
            __syncthreads();
        } 
        result += block_result;//  * as[row + inner_dim_idx/BLOCK * m] * bs[col/BLOCK + inner_dim_idx/BLOCK * sn];
        __syncthreads();
    } 
    c[row* n + col] = result;
}


__global__ void custom_kernel_bak_bak(const float* A, const float* B, const float* as, const float* bs, float* C, int m, int n, int k) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int cx = bx * TILE_WIDTH + tx; 
    int cy = by * TILE_WIDTH + ty; 

    if (cx >= m || cy >= n) {
        return;
    }

    float C_val = 0.0;

    __shared__ float A_tile[TILE_WIDTH][TILE_WIDTH];
    __shared__ float B_tile[TILE_WIDTH][TILE_WIDTH];

    for (int i = 0; i < k/TILE_WIDTH; i++) { 
        // fill the tile with a row of A and a row of B
        // we need row cx of A and row cy of B
        // fill one row of the tile with one column slice
        A_tile[ty][tx] = A[cx + (i * TILE_WIDTH + ty ) * m]; //A[tx][ty];
        B_tile[ty][tx] = B[tx + (i * TILE_WIDTH + cy ) * m];
        __syncthreads();

//        for (int q = 0; q < TILE_WIDTH; q++) {
//            for (int w = 0; w < TILE_WIDTH; w++) {
//                A_tile[q][w] == A[];
//
//            }
//        }

        for (int j = 0; j < TILE_WIDTH; j++) {
            //C_val += A_tile[j][ty] * B_tile[j][tx];
            C_val += A_tile[j][tx] * B_tile[j][ty];
        }
        __syncthreads();
    }

    C[cx * n + cy] = C_val;


}

__global__ void custom_kernel(const float* A, const float* B, const float* as, const float* bs, float* C, int m, int n, int k) {
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int cx = bx * TILE_WIDTH + tx; 
    int cy = by * TILE_WIDTH + ty; 

    int lane_id = (threadIdx.y * blockDim.x + threadIdx.x);

    float A_val = 0.0;
    float B_val = 0.0;
    float C_val = 0.0;

    //if (cx == 0 && cy == 0) {
    //    printf("%d %d %d %d", blockDim.x, blockDim.y, threadIdx.x, threadIdx.y);
    //}
    for (int i = 0; i < k; i += blockDim.x) {
        if (lane_id < 16) {
            A_val = A[cx + m * (i + lane_id%8)];    
        } else {
            B_val = B[cy + n * (i + lane_id%8)];    
        }
            if (i == 0 && cx == 0 && cy == 0) {
                
                //for (int A_tile_idx = 0; A_tile_idx < 16; A_tile_idx++) {
                //    printf("%f ", __shfl_sync(0xffffffff, A_val, A_tile_idx));
                //} 

                printf("\n");
                
                for (int A_row = 0; A_row < 2; A_row++) {
                    for (int A_col = 0; A_col < 8; A_col++) {
                        printf("%f ", A[A_row + A_col * m]);
                }
                printf("\n");
            }
            }

        for (int j = 0; j < blockIdx.x; j++) {
            //if (i == 0) {
            //    printf("%f %f \n", __shfl_sync(0xffffffff, A_val, j) , __shfl_sync(0xffffffff, B_val, j + 16));
            //}
            C_val += __shfl_sync(0xffffffff, A_val, j) * __shfl_sync(0xffffffff, B_val, j + 16);
        }

    }

    C[cx * n + cy] = C_val;
}

int main() {

      int m = 128;
      float diff = 0.0;
      int runs = 10;
      for (int i = 0; i < runs; i++) {
         diff += compare_mat_mul_kernels(mat_mul_ref, custom_kernel_bak_bak, m, m, m, true, dim3(m/TILE_WIDTH,m/TILE_WIDTH ), dim3(TILE_WIDTH, TILE_WIDTH), dim3(m/8, m/4), dim3(8, 4)); 
      }
      printf("\n%f\n", diff/runs);
      diff = 0.0;
     // for (int i = 0; i < runs; i++) {
     //     diff += compare_mat_mul_kernels(mat_mul_ref, custom_kernel, m, m, m, false, dim3(m/TILE_WIDTH,m/TILE_WIDTH ), dim3(TILE_WIDTH, TILE_WIDTH), dim3(m/TILE_WIDTH, m/TILE_WIDTH), dim3(TILE_WIDTH, TILE_WIDTH)); 
     // }
     // printf("\n%f\n", diff/runs);
}

